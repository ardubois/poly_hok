#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <malloc.h>

#define _bitsperpixel 32
#define _planes 1
#define _compression 0
#define _xpixelpermeter 0x13B //0x130B //2835 , 72 DPI
#define _ypixelpermeter 0x13B//0x130B //2835 , 72 DPI

#pragma pack(push,1)
typedef struct{
    uint8_t signature[2];
    uint32_t filesize;
    uint32_t reserved;
    uint32_t fileoffset_to_pixelarray;
} fileheader;
typedef struct{
    uint32_t dibheadersize;
    uint32_t width;
    uint32_t height;
    uint16_t planes;
    uint16_t bitsperpixel;
    uint32_t compression;
    uint32_t imagesize;
    uint32_t ypixelpermeter;
    uint32_t xpixelpermeter;
    uint32_t numcolorspallette;
    uint32_t mostimpcolor;
} bitmapinfoheader;
typedef struct {
    fileheader fileheader;
    bitmapinfoheader bitmapinfoheader;
} bitmap;
#pragma pack(pop)

void genBpm (int height, int width, float *pixelbuffer_f) {
    uint32_t pixelbytesize = height*width*_bitsperpixel/8;
    uint32_t  _filesize =pixelbytesize+sizeof(bitmap);
    FILE *fp = fopen("julia.bmp","wb");
    bitmap *pbitmap  = (bitmap*)calloc(1,sizeof(bitmap));

    int buffer_size = height*width*4;
    uint8_t *pixelbuffer = (uint8_t*)malloc(buffer_size);

    for(int i = 0; i<buffer_size;i++)
    {
     pixelbuffer[i]= (uint8_t) pixelbuffer_f[i];
    }


    //strcpy(pbitmap->fileheader.signature,"BM");
    pbitmap->fileheader.signature[0] = 'B';
    pbitmap->fileheader.signature[1] = 'M';
    pbitmap->fileheader.filesize = _filesize;
    pbitmap->fileheader.fileoffset_to_pixelarray = sizeof(bitmap);
    pbitmap->bitmapinfoheader.dibheadersize =sizeof(bitmapinfoheader);
    pbitmap->bitmapinfoheader.width = width;
    pbitmap->bitmapinfoheader.height = height;
    pbitmap->bitmapinfoheader.planes = _planes;
    pbitmap->bitmapinfoheader.bitsperpixel = _bitsperpixel;
    pbitmap->bitmapinfoheader.compression = _compression;
    pbitmap->bitmapinfoheader.imagesize = pixelbytesize;
    pbitmap->bitmapinfoheader.ypixelpermeter = _ypixelpermeter ;
    pbitmap->bitmapinfoheader.xpixelpermeter = _xpixelpermeter ;
    pbitmap->bitmapinfoheader.numcolorspallette = 0;
    fwrite (pbitmap, 1, sizeof(bitmap),fp);
    //memset(pixelbuffer,pixel,pixelbytesize);
    fwrite(pixelbuffer,1,pixelbytesize,fp);
    fclose(fp);
    free(pbitmap);
    free(pixelbuffer);
}

int main( int argc, char const *argv[] ) {

    int usr_value = atoi(argv[1]);
   
    int height = usr_value;
    int width  = usr_value;
    int DIM = usr_value;
    int size_array = height*width*4*sizeof(int);
    hipError_t j_error;
    
    //int pixelbytesize=  height*width*_bitsperpixel/8;
    //printf(" pixel byte size %lu\n",pixelbytesize);
   
    float time;
    hipEvent_t start, stop;   
    hipEventCreate(&start) ;
    hipEventCreate(&stop) ;
    hipEventRecord(start, 0) ;


     int *h_pixelbuffer = (int*)malloc(size_array);
     int *d_pixelbuffer;

     ////////
    hipMalloc( (void**)&d_pixelbuffer, size_array);
    j_error = hipGetLastError();
    if(j_error != hipSuccess) printf("Error 1: %s\n", hipGetErrorString(j_error));
    ////////

    
    ////////////////////
    dim3 grid(DIM,DIM);

   // int (*f)(float*,int,int,int) = (int (*)(float*,int,int,int)) get_julia_function_ptr();

    mapgen2D_xy_1para_noret_ker<<<grid, 1>>>(d_pixelbuffer,DIM,DIM,f);
    
    j_error = hipGetLastError();
    if(j_error != hipSuccess) printf("Error 3: %s\n", hipGetErrorString(j_error));
  ////////



    hipMemcpy(h_pixelbuffer, d_pixelbuffer, size_array, hipMemcpyDeviceToHost); // return results 
    j_error = hipGetLastError();
    if(j_error != hipSuccess) printf("Error 7: %s\n", hipGetErrorString(j_error));



    hipEventRecord(stop, 0) ;
    hipEventSynchronize(stop) ;
    hipEventElapsedTime(&time, start, stop) ;

    printf("CUDA\t%d\t%3.1f\n", usr_value,time);
    
    //genBpm(height,width,h_pixelbuffer);
   
    free(h_pixelbuffer);
    hipFree(d_pixelbuffer);
}



